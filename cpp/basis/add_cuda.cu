// https://developer.nvidia.com/blog/even-easier-introduction-cuda/


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
// Kernel function to add the elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  // spread the computation to threads and blocks
  int index = blockIdx.x*blockDim.x + threadIdx.x;
  int stride = blockDim.x*gridDim.x; //threads in the grid; gridDim.x->how many blocks; blockDim.x ->how many threads
  for (int i = index; i < n; i+=stride) // grid-stride loop: thread 0 compute 0, 0+stride, 0+2*stride, ...
    y[i] = x[i] + y[i];
}

int main(void)
{
  int N = 1<<20;
  float *x, *y;

  // Allocate Unified Memory – accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));

  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }

  int blockSize = 256; // threads per block
  int numBlock = (N+blockSize-1)/blockSize; //round up N in case N is not a multiple of blockSize

  int numSMs;
  hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);
  numBlock = 32*numSMs;
  std::cout << "Number of blocks: "<< numBlock << std::endl;
  // Run kernel on 1M elements on the GPU
  add<<<numBlock, blockSize>>>(N, x, y);
  // <<blocks, thread>>
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;

  // Free memory
  hipFree(x);
  hipFree(y);
  
  return 0;
}